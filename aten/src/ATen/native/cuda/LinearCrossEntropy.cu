#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/Dispatch.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/linear_cross_entropy_native.h>
#include <ATen/ops/linear.h>
#include <ATen/ops/cross_entropy_loss.h>
#include <ATen/ops/zeros.h>
#include <ATen/ops/logical_and.h>
#include <ATen/ops/ge.h>
#include <ATen/ops/lt.h>
#include <ATen/ops/where.h>
#include <ATen/ops/sub.h>
#include <ATen/ops/ne.h>
#include <ATen/ops/sum.h>
#include <ATen/ops/div.h>
#endif

namespace at::native {

// Forward declarations
Tensor batch_chunking_cuda(
    const Tensor& input,
    const Tensor& weight,
    const Tensor& target,
    const std::optional<Tensor>& bias_opt,
    int64_t reduction,
    int64_t ignore_index,
    double label_smoothing);

// Utility functions for strategy selection heuristics
// Based on empirical analysis from memory profiling (Milestone 2) and CPU implementation
inline bool should_use_vocab_chunking_cuda(int64_t vocab_size, int64_t batch_size) {
  // Use vocabulary chunking for large vocabularies (LLM training scenarios)
  // Threshold based on proven CPU implementation and memory constraints
  return vocab_size > 8192;
}

inline bool should_use_batch_chunking_cuda(int64_t vocab_size, int64_t batch_size) {
  // Use batch chunking for large batches with moderate vocabularies (fine-tuning scenarios)
  // Mirrors CPU implementation logic: batch_large && !vocab_large
  const int64_t batch_chunk_threshold = 1024;
  const int64_t vocab_chunk_threshold = 8192;
  return batch_size > batch_chunk_threshold && vocab_size <= vocab_chunk_threshold;
}

// Apply final reduction based on reduction mode
// Handles mean/sum reduction consistently with PyTorch cross_entropy behavior
Tensor apply_reduction_cuda(const Tensor& total_loss, int64_t valid_count, int64_t reduction) {
  if (reduction == Reduction::Mean) {
    if (valid_count > 0) {
      return at::div(total_loss, valid_count);
    } else {
      return total_loss; // Will be 0 if no valid samples
    }
  } else if (reduction == Reduction::Sum) {
    return total_loss;
  } else { // Reduction::None
    TORCH_CHECK(false, "Reduction::None not supported for vocabulary chunking yet");
  }
}

// CUDA vocabulary chunking implementation
// Based on established approaches from PyTorch Issue #124480 and proven CPU implementation
// Uses cuBLAS for matrix operations and cuDNN for cross-entropy (library reuse strategy)
Tensor vocab_chunking_cuda(
    const Tensor& input,
    const Tensor& weight,
    const Tensor& target,
    const std::optional<Tensor>& bias_opt,
    int64_t reduction,
    int64_t ignore_index,
    double label_smoothing) {
  
  // Input validation - ensure all tensors are on CUDA device
  TORCH_CHECK(input.is_cuda(), "linear_cross_entropy_cuda: input must be CUDA tensor");
  TORCH_CHECK(weight.is_cuda(), "linear_cross_entropy_cuda: weight must be CUDA tensor");
  TORCH_CHECK(target.is_cuda(), "linear_cross_entropy_cuda: target must be CUDA tensor");
  TORCH_CHECK(input.device() == weight.device(), 
              "linear_cross_entropy_cuda: input and weight must be on same device");
  TORCH_CHECK(input.device() == target.device(), 
              "linear_cross_entropy_cuda: input and target must be on same device");
  
  // Validate bias if provided
  if (bias_opt.has_value()) {
    const auto& bias = bias_opt.value();
    TORCH_CHECK(bias.is_cuda(), "linear_cross_entropy_cuda: bias must be CUDA tensor");
    TORCH_CHECK(bias.device() == input.device(), 
                "linear_cross_entropy_cuda: bias must be on same device as input");
  }
  
  // Flatten multi-dimensional inputs (mirrors proven CPU implementation)
  // Handles both 2D [batch, hidden] and 3D [batch, seq, hidden] inputs
  const auto input_flat = input.view({-1, input.size(-1)});  // [N, H] where N = batch * seq_len
  const auto target_flat = target.view({-1});                // [N] flattened targets
  
  const int64_t vocab_size = weight.size(0);
  const int64_t chunk_size = 4096;  // Same optimal chunk size as CPU implementation
  const int64_t num_chunks = (vocab_size + chunk_size - 1) / chunk_size;
  
  // Initialize accumulators on GPU
  auto total_loss = at::zeros({}, input.options());
  int64_t valid_count = 0;
  
  // Process each vocabulary chunk using native CUDA operations
  // This mirrors the CPU algorithm but uses cuBLAS/cuDNN instead of CPU operations
  for (int64_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx) {
    const int64_t start_idx = chunk_idx * chunk_size;
    const int64_t end_idx = std::min(start_idx + chunk_size, vocab_size);
    
    // Extract vocabulary chunk (key insight from Issue #124480)
    // Process weight matrix in vocabulary dimension to avoid large logit tensor
    auto weight_chunk = weight.slice(0, start_idx, end_idx);  // [chunk_vocab_size, hidden_dim]
    
    // Extract corresponding bias chunk if bias is provided
    std::optional<Tensor> bias_chunk;
    if (bias_opt.has_value()) {
      bias_chunk = bias_opt->slice(0, start_idx, end_idx);
    }
    
    // Use cuBLAS-optimized matrix multiplication (at::linear uses cuBLAS internally)
    // Only materializes [N, chunk_size] instead of [N, vocab_size] tensor - key memory savings
    auto logits_chunk = at::linear(input_flat, weight_chunk.t(), bias_chunk);  // [N, chunk_vocab_size]
    
    // Create boolean mask for samples with targets in current vocabulary chunk
    // This implements selective loss computation from vocabulary chunking theory
    auto target_mask = at::logical_and(
        at::ge(target_flat, start_idx),   // target >= start_idx
        at::lt(target_flat, end_idx)      // target < end_idx
    );
    
    // Only process chunks that contain relevant targets (optimization)
    if (target_mask.any().item().toBool()) {
      // Adjust target indices to chunk-local indices (0-based within chunk)
      auto target_chunk = at::sub(target_flat, start_idx);
      
      // Apply ignore_index mask (convert out-of-range targets to ignore_index)
      target_chunk = at::where(target_mask, target_chunk, ignore_index);
      
      // Use cuDNN-optimized cross-entropy computation
      // This leverages the same optimized kernels used by standard PyTorch cross_entropy
      auto chunk_loss = at::cross_entropy_loss(
          logits_chunk,           // [N, chunk_vocab_size] logits
          target_chunk,           // [N] chunk-local target indices
          /*weight=*/std::nullopt, // no class weights (can be added in future)
          Reduction::Sum,         // sum within chunk, apply final reduction later
          ignore_index,           // ignore specified index
          label_smoothing         // label smoothing factor
      );
      
      // Accumulate results across chunks (GPU-native operations)
      total_loss += chunk_loss;
      valid_count += target_mask.sum().item().toLong();
    }
  }
  
  // Apply final reduction (mean/sum) based on accumulated results
  return apply_reduction_cuda(total_loss, valid_count, reduction);
}

// Naive CUDA implementation for small vocabularies
// Uses standard PyTorch operations without chunking (no memory optimization needed)
Tensor naive_linear_cross_entropy_cuda(
    const Tensor& input,
    const Tensor& weight,
    const Tensor& target,
    const std::optional<Tensor>& bias_opt,
    int64_t reduction,
    int64_t ignore_index,
    double label_smoothing) {
  
  // Standard linear + cross_entropy operations for small models
  // This path avoids chunking overhead when memory savings aren't needed
  auto logits = at::linear(input, weight.t(), bias_opt);
  
  // Reshape tensors for cross_entropy compatibility
  // cross_entropy expects [N, C] logits and [N] targets
  auto logits_flat = logits.view({-1, logits.size(-1)});  // [N, C]
  auto target_flat = target.view({-1});                   // [N]
  
  return at::cross_entropy_loss(
      logits_flat, target_flat,
      /*weight=*/std::nullopt,
      reduction,
      ignore_index,
      label_smoothing
  );
}

// Main CUDA implementation entry point
// Implements Phase 4a: CUDA Vocabulary Chunking (replaces CPU delegation)
Tensor linear_cross_entropy_cuda(
    const Tensor& input,
    const Tensor& weight,
    const Tensor& target,
    const std::optional<Tensor>& bias_opt,
    int64_t reduction,
    int64_t ignore_index,
    double label_smoothing,
    c10::string_view chunking_strategy) {
  
  // Input dimension validation (matches CPU implementation requirements)
  TORCH_CHECK(input.dim() >= 2, "linear_cross_entropy_cuda: input must have at least 2 dimensions");
  TORCH_CHECK(weight.dim() == 2, "linear_cross_entropy_cuda: weight must be 2-dimensional");
  TORCH_CHECK(input.size(-1) == weight.size(1), 
              "linear_cross_entropy_cuda: input.size(-1) must match weight.size(1)");
  
  const int64_t vocab_size = weight.size(0);
  const int64_t batch_size = input.view({-1, input.size(-1)}).size(0);
  
  // Strategy selection based on input characteristics
  if (chunking_strategy == "vocab" || 
      (chunking_strategy == "auto" && should_use_vocab_chunking_cuda(vocab_size, batch_size))) {
    // Use vocabulary chunking for large vocabularies (LLM training scenarios)
    return vocab_chunking_cuda(input, weight, target, bias_opt, reduction, ignore_index, label_smoothing);
  } else if (chunking_strategy == "batch" ||
             (chunking_strategy == "auto" && should_use_batch_chunking_cuda(vocab_size, batch_size))) {
    // Use batch chunking for large batch sizes with moderate vocabularies (fine-tuning scenarios)
    // Phase 4c: Native CUDA batch chunking implementation
    return batch_chunking_cuda(input, weight, target, bias_opt, reduction, ignore_index, label_smoothing);
  } else {
    // Use naive implementation for small models (no chunking overhead)
    return naive_linear_cross_entropy_cuda(input, weight, target, bias_opt, reduction, ignore_index, label_smoothing);
  }
}

// CUDA batch chunking implementation for Phase 4c
// Mirrors CPU batch_chunking_cpu() algorithm using cuBLAS/cuDNN operations
// Inspired by Liger Kernel approach but maintains PyTorch library reuse strategy
Tensor batch_chunking_cuda(
    const Tensor& input,
    const Tensor& weight,
    const Tensor& target,
    const std::optional<Tensor>& bias_opt,
    int64_t reduction,
    int64_t ignore_index,
    double label_smoothing) {
  
  // Input validation - ensure all tensors are on CUDA device
  TORCH_CHECK(input.is_cuda(), "batch_chunking_cuda: input must be CUDA tensor");
  TORCH_CHECK(weight.is_cuda(), "batch_chunking_cuda: weight must be CUDA tensor");
  TORCH_CHECK(target.is_cuda(), "batch_chunking_cuda: target must be CUDA tensor");
  TORCH_CHECK(input.device() == weight.device(), 
              "batch_chunking_cuda: input and weight must be on same device");
  TORCH_CHECK(input.device() == target.device(), 
              "batch_chunking_cuda: input and target must be on same device");
  
  // Validate bias if provided
  if (bias_opt.has_value()) {
    const auto& bias = bias_opt.value();
    TORCH_CHECK(bias.is_cuda(), "batch_chunking_cuda: bias must be CUDA tensor");
    TORCH_CHECK(bias.device() == input.device(), 
                "batch_chunking_cuda: bias must be on same device as input");
  }
  
  // Flatten multi-dimensional inputs for processing (mirrors CPU implementation)
  // This allows handling both 2D [batch, hidden] and 3D [batch, seq, hidden] inputs
  const auto input_flat = input.view({-1, input.size(-1)});  // [N, H] where N = batch * seq_len
  const auto target_flat = target.view({-1});                // [N] flattened targets
  
  const int64_t batch_size = input_flat.size(0);
  const int64_t chunk_size = 1024;  // Same optimal chunk size as CPU implementation (empirically validated)
  
  // Early exit if batch is too small for chunking (mirrors CPU logic)
  // Use naive implementation to avoid chunking overhead
  if (batch_size <= chunk_size) {
    auto logits = at::linear(input_flat, weight.t(), bias_opt);
    return at::cross_entropy_loss(logits, target_flat, /*weight=*/std::nullopt, 
                                 reduction, ignore_index, label_smoothing);
  }
  
  const int64_t num_chunks = (batch_size + chunk_size - 1) / chunk_size;
  
  // Initialize accumulators on GPU (same pattern as vocab_chunking_cuda)
  auto total_loss = at::zeros({}, input.options());
  int64_t valid_count = 0;
  
  // Process input in batch chunks to avoid materializing large logit tensors
  // Each chunk computes: [chunk_size, hidden] @ [hidden, vocab] -> [chunk_size, vocab]
  // This is the key operation: smaller batch × full vocab instead of full batch × full vocab
  for (int64_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx) {
    const int64_t start_idx = chunk_idx * chunk_size;
    const int64_t end_idx = std::min(start_idx + chunk_size, batch_size);
    
    // Skip empty chunks (edge case safety, mirrors CPU implementation)
    if (start_idx >= end_idx) continue;
    
    // Extract batch chunk - memory efficient slicing using CUDA tensors
    // These operations are optimized by PyTorch for contiguous memory access
    auto input_chunk = input_flat.slice(0, start_idx, end_idx);   // [actual_chunk_size, H]
    auto target_chunk = target_flat.slice(0, start_idx, end_idx); // [actual_chunk_size]
    
    // Use cuBLAS-optimized matrix multiplication (at::linear uses cuBLAS internally)
    // This leverages NVIDIA's highly optimized GEMM kernels
    // Key insight: smaller batch × full vocab instead of full batch × full vocab
    auto logits_chunk = at::linear(input_chunk, weight.t(), bias_opt); // [actual_chunk_size, vocab_size]
    
    // Use cuDNN-optimized cross-entropy computation (at::cross_entropy_loss uses cuDNN)
    // This leverages NVIDIA's optimized loss computation kernels
    auto chunk_loss = at::cross_entropy_loss(
        logits_chunk,           // [actual_chunk_size, vocab_size] logits
        target_chunk,           // [actual_chunk_size] targets  
        /*weight=*/std::nullopt, // no class weights (can be added in future)
        Reduction::Sum,         // sum within chunk, apply final reduction later
        ignore_index,           // ignore specified index
        label_smoothing         // label smoothing factor
    );
    
    // Accumulate results across chunks (GPU-native operations)
    total_loss += chunk_loss;
    
    // Count valid samples (excluding ignore_index) for mean reduction
    // This mirrors the CPU implementation exactly
    auto valid_mask = at::ne(target_chunk, ignore_index);
    valid_count += valid_mask.sum().item().toLong();
  }
  
  // Apply final reduction (mean/sum) based on accumulated results
  // Uses the same reduction logic as vocabulary chunking
  return apply_reduction_cuda(total_loss, valid_count, reduction);
}

} // namespace at::native